#include "hip/hip_runtime.h"
#include "embed.h"
#include "grouping.h"

__global__ void embed(unsigned char *data, const int data_size, unsigned char *secrets, const int num_secret, cosets* sub_g, int start)
{
    printf("hello\n");
    unsigned char temp[7];
    uint8_t u=0, v=0; 
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int count = start+threadId/7;
    int start_pos = start + threadId;
    int stride = blockDim.x * gridDim.x;
    printf("hello3\n");
    for (int n=0; n < num_secret/stride; n++) {
	uint8_t i = threadId%7;
        temp[i] = secrets[count+i];   
	__syncthreads();
        u = temp[2] * 8
          + temp[4] * 4
          + temp[5] * 2
          + temp[6] * 1;
          
        v = temp[0] * 4
          + temp[1] * 2
          + temp[3] * 1;
    	printf("hello2\n");
        if (sub_g[u*8+v].bit[i] == 1){
            data[i + start_pos] |= (unsigned char)1; //0b00000001
        }
        else{
            data[i + start_pos] &= (unsigned char)254; //0b11111110
        }
        start_pos += stride;
    }
    printf("%d %d %d\n", threadId, u, v);
}

__host__ void remain_embed(unsigned char *data, const int data_size, unsigned char *secrets, const int num_secret, cosets* sub_g)
{
    unsigned char pixcel_mask = 0;
    const uint8_t remain = (num_secret) % 7;
    unsigned char data_mask = 255;
    unsigned char temp[7];
    uint8_t u, v; 
    for (uint8_t i = 0; i < 7; i++){
        temp[i] = 0;
        if (i < remain){
            temp[i] = secrets[i + num_secret - remain];
        }   
    }
    u = temp[2] * 8
      + temp[4] * 4
      + temp[5] * 2
      + temp[6] * 1;
    
    v = temp[0] * 4
      + temp[1] * 2
      + temp[3] * 1;
      
    for (uint8_t i = 0; i < remain; i++){
        if (sub_g[u*8+v].bit[i] == 1){
            data[i + num_secret - remain] |= (unsigned char)1; //0b0000001
        }
        else{
            data[i + num_secret - remain] &= (unsigned char)254; //0b1111110
        }
    }
    // extra n bits replace the smallest n bits of the last pixel 
    for (uint8_t i = remain; i < 7; i++){
        pixcel_mask |= sub_g[u*8+v].bit[i];
        if (i < 6){
            pixcel_mask <<= 1;
        }
        data_mask <<= 1;
    }
    data[num_secret] &= data_mask;
    data[num_secret] |= pixcel_mask;

}
