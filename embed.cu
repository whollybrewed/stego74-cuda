#include "hip/hip_runtime.h"
#include "embed.h"
#include "grouping.h"

__global__ void embed(unsigned char *data, 
                      const int data_size, 
                      unsigned char *secrets, 
                      const int num_secret, 
                      cosets* sub_g, 
                      int start)
{
    __shared__ unsigned char temp[32][7];
    int u=0, v=0; 
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int start_pos = start + (threadId/7)*7;
    int stride = blockDim.x * gridDim.x;
    for (int n=0; n < num_secret/stride; n++) {
        int i = threadId%7;
        int small_thread = (threadId%blockDim.x)/7;
        temp[small_thread][i] = secrets[start_pos + i];   
        __syncthreads();
        u = temp[small_thread][2] * 8
          + temp[small_thread][4] * 4
          + temp[small_thread][5] * 2
          + temp[small_thread][6] * 1;
            
        v = temp[small_thread][0] * 4
          + temp[small_thread][1] * 2
          + temp[small_thread][3] * 1;
        if (sub_g[u*8+v].bit[i] == 1){
            data[i + start_pos] |= (unsigned char)1; //0b00000001
        }
        else{
            data[i + start_pos] &= (unsigned char)254; //0b11111110
        }
        __syncthreads();
        start_pos += stride;
    }
}

__host__ void remain_embed(unsigned char *data, 
                           const int data_size, 
                           unsigned char *secrets, 
                           const int num_secret, 
                           cosets* sub_g)
{
    unsigned char pixcel_mask = 0;
    const uint8_t remain = (num_secret) % 7;
    unsigned char data_mask = 255;
    unsigned char temp[7];
    uint8_t u, v; 
    for (uint8_t i = 0; i < 7; i++){
        temp[i] = 0;
        if (i < remain){
            temp[i] = secrets[i + num_secret - remain];
        }   
    }
    u = temp[2] * 8
      + temp[4] * 4
      + temp[5] * 2
      + temp[6] * 1;
    
    v = temp[0] * 4
      + temp[1] * 2
      + temp[3] * 1;
      
    for (uint8_t i = 0; i < remain; i++){
        if (sub_g[u*8+v].bit[i] == 1){
            data[i + num_secret - remain] |= (unsigned char)1; //0b0000001
        }
        else{
            data[i + num_secret - remain] &= (unsigned char)254; //0b1111110
        }
    }
    // extra n bits replace the smallest n bits of the last pixel 
    for (uint8_t i = remain; i < 7; i++){
        pixcel_mask |= sub_g[u*8+v].bit[i];
        if (i < 6){
            pixcel_mask <<= 1;
        }
        data_mask <<= 1;
    }
    data[num_secret] &= data_mask;
    data[num_secret] |= pixcel_mask;
}
