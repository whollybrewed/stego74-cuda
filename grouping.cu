#include "hip/hip_runtime.h"
#include "grouping.h"

__global__ void grouping(cosets *d_sub_g)
{
    unsigned char matrix_h[3][7] =
    {
        {1, 1, 0, 1, 1, 0, 0},
        {1, 0, 1, 1, 0, 1, 0},
        {0, 1, 1, 1, 0, 0, 1}
    };

    unsigned char syndrome[3];
    unsigned char entry[7];
    int u, v;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int j = 0; j < 7; j++){
        // store the decimal value in the binary format of one bit per entry
        entry[j] = (((idx >> j) & 1) == 0 ? 0 : 1);
    }
    __syncthreads();

    u = entry[2] * 8
      + entry[4] * 4
      + entry[5] * 2
      + entry[6] * 1;
   
    for (int i = 0; i < 3; i++){
        for (int j = 0; j < 7; j++){
            if (j == 0)
                syndrome[i] = entry[j] * matrix_h[i][j];
            else	
                syndrome[i] = syndrome[i] ^ (entry[j] * matrix_h[i][j]);
        }	
    }

    v = syndrome[0] * 4
      + syndrome[1] * 2
      + syndrome[2] * 1;

    for (int i = 0; i < 7; i++){
        d_sub_g[u * 8 + v].bit[i] = entry[i];
    }
}