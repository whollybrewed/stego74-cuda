#include <stdio.h>
#include <stdlib.h>
#include "bmp_parser.h"
#include "embed.h"
#include "decoder.h"

int main(int argc, char* argv[])
{
    // bmp reader
    struct BmpParser encoder;
    ReadFile("photo/fruit.bmp", &encoder);
    
    // read message from txt file
    int secret_size = (encoder.width*encoder.height-1)*8;
    char* string = (char*)malloc((encoder.width*encoder.height-1)*sizeof(unsigned char)+1);
    ReadTxt(argv[1], string, encoder.width*encoder.height-1);

    // stego encoder
    unsigned char* bits = (unsigned char*)malloc(secret_size*sizeof(unsigned char));
    StringToBits(string, bits);

    cosets *d_sub_g;
    cosets host_sub_g[128];
    dim3 BlockSize(128);
    dim3 GridSize(1);
    int sub_g_size = 128 * sizeof(cosets);
    hipMalloc((void**)&d_sub_g, sub_g_size);
    grouping<<<GridSize, BlockSize>>>(d_sub_g);
    hipMemcpy(host_sub_g, d_sub_g, sub_g_size, hipMemcpyDeviceToHost);
    //hipBindTexture(0, d_tex, d_sub_g, sub_g_size);
    //texture <cosets, 1, hipReadModeElementType> d_tex;
    unsigned char *data_cu, *secret_cu;
    int tile_width = 224;
    int remain = secret_size % 7;
    int b_remain = (secret_size-remain)%tile_width;
    dim3 dimBlock(tile_width);
    dim3 dimGrid(secret_size/tile_width);
    hipMalloc((void**)&data_cu, secret_size+1);
    hipMalloc((void**)&secret_cu, secret_size);
    hipMemcpy(data_cu, encoder.data, encoder.data_size, hipMemcpyHostToDevice);
    hipMemcpy(secret_cu, bits, secret_size, hipMemcpyHostToDevice);
    hipStream_t stream1, stream2;
    //hipStreamCreate(&stream1);
    //hipStreamCreate(&stream2);
    //embed<<<dimGrid, dimBlock, 0, stream1>>> (encoder.data, encoder.data_size, bits, secret_size-remain-b_remain, d_sub_g, 0);
    //embed<<<1, b_remain, 0, stream2>>> (encoder.data, encoder.data_size, bits, secret_size, d_sub_g, secret_size - remain - b_remain);
    embed<<<dimGrid, dimBlock>>> (encoder.data, encoder.data_size, bits, secret_size-remain-b_remain, d_sub_g, 0);
    embed<<<1, b_remain>>> (encoder.data, encoder.data_size, bits, secret_size, d_sub_g, secret_size - remain - b_remain);
    remain_embed(encoder.data, encoder.data_size, bits, secret_size, host_sub_g);
    hipMemcpy(encoder.data, data_cu, encoder.data_size-remain, hipMemcpyDeviceToHost);
    printf("\nAfter encode\n");
    OutputFile("photo/encode.bmp", &encoder);
    free(bits);
    free(string);
    hipFree(data_cu);
    hipFree(secret_cu);
    // stego decoder
    struct BmpParser decoder;
    ReadFile("photo/encode.bmp", &decoder);
    char* message = (char*)malloc((secret_size/8)*sizeof(char)+1);
    decode(decoder.data, secret_size, message);
    printf("message = %s\n", message);

    // output txt file
    OutputTxt("message.txt", message);
    free(message);
    return 0;
}
