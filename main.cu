#include <stdio.h>
#include <stdlib.h>
#include "bmp_parser.h"
#include "embed.h"
#include "decoder.h"

int main(int argc, char* argv[])
{
    // bmp reader
    struct BmpParser encoder;
    ReadFile("photo/fruit.bmp", &encoder);
    
    // read message from txt file
    int secret_size = (encoder.width*encoder.height-1)*8;
    char* string = (char*)malloc((encoder.width*encoder.height-1)*sizeof(unsigned char)+1);
    ReadTxt(argv[1], string, encoder.width*encoder.height-1);

    // stego encoder
    unsigned char* bits = (unsigned char*)malloc(secret_size*sizeof(unsigned char));
    StringToBits(string, bits);

    //debug purpose
    cosets temp_sub_g[16 * 8];

    cosets *d_sub_g;
    dim3 BlockSize(128);
    dim3 GridSize(1);
    int sub_g_size = 128 * sizeof(cosets);
    hipMalloc((void**)&d_sub_g, sub_g_size);
    texture <cosets, 1, hipReadModeElementType> d_tex;
    grouping<<<GridSize, BlockSize>>>(d_sub_g);
    hipMemcpy(temp_sub_g, d_sub_g, sub_g_size, hipMemcpyDeviceToHost);
    hipBindTexture(0, d_tex, d_sub_g, sub_g_size);

    embed(encoder.data, encoder.data_size, bits, secret_size, temp_sub_g);
    printf("\nAfter encode\n");
    OutputFile("photo/encode.bmp", &encoder);
    free(bits);
    free(string);

    // stego decoder
    struct BmpParser decoder;
    ReadFile("photo/encode.bmp", &decoder);
    char* message = (char*)malloc((secret_size/8)*sizeof(char)+1);
    decode(decoder.data, secret_size, message);
    printf("message = %s\n", message);

    // output txt file
    OutputTxt("message.txt", message);
    free(message);
    return 0;
}