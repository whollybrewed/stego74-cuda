#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "bmp_parser.h"
#include "embed.h"
#include "decoder.h"

int main(int argc, char* argv[])
{
    //timer
    clock_t start, end;
    double embed_time, decode_time;

    // bmp reader
    struct BmpParser encoder;
    ReadFile(argv[2], &encoder);
    
    // read message from txt file
    int secret_size = (encoder.width*encoder.height-1);
    int string_size = secret_size/8;
    char* string = (char*)malloc(string_size*sizeof(unsigned char)+1);
    ReadTxt(argv[1], string, string_size);

    // stego encoder
    unsigned char* bits = (unsigned char*)malloc(secret_size*sizeof(unsigned char));
    StringToBits(string, bits);
    printf("Embedding...\n");
    start = clock();
    embed(encoder.data, encoder.data_size, bits, secret_size);
    end = clock();
    embed_time = ((double) (end - start)) / CLOCKS_PER_SEC;
    OutputFile("photo/encode.bmp", &encoder);
    printf("Output embedded image\n");
    free(bits);
    free(string);

    // stego decoder
    struct BmpParser decoder;
    ReadFile("photo/encode.bmp", &decoder);
    char* message = (char*)malloc((secret_size/8)*sizeof(char)+1);
    printf("Decoding...\n");
    start = clock();
    decode(decoder.data, secret_size, message);
    end = clock();
    decode_time = ((double) (end - start)) / CLOCKS_PER_SEC;
    OutputTxt("message.txt", message);
    printf("Output decoded message\n");
    free(message);
    printf("==================================================\n");
    printf("embed time: %f ms\n", embed_time/1000);
    printf("decode time: %f ms\n", decode_time/1000);
    return 0;
}
