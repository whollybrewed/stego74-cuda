#include <stdio.h>
#include <stdlib.h>
#include "bmp_parser.h"
#include "embed.h"
#include "decoder.h"

int main(int argc, char* argv[])
{
    // bmp reader
    struct BmpParser encoder;
    ReadFile(argv[2], &encoder);
    
    // read message from txt file
    int secret_size = (encoder.width*encoder.height-1);
    int string_size = secret_size/8;
    char* string = (char*)malloc(string_size*sizeof(unsigned char)+1);
    ReadTxt(argv[1], string, string_size);
    // stego encoder
    unsigned char* bits = (unsigned char*)malloc(secret_size*sizeof(unsigned char));
    StringToBits(string, bits);
    printf("After String to bit\n");
    cosets *d_sub_g;
    cosets host_sub_g[128];
    dim3 BlockSize(128);
    dim3 GridSize(1);
    int sub_g_size = 128 * sizeof(cosets);
    hipMalloc((void**)&d_sub_g, sub_g_size);
    grouping<<<GridSize, BlockSize>>>(d_sub_g);
    hipMemcpy(host_sub_g, d_sub_g, sub_g_size, hipMemcpyDeviceToHost);
    //hipBindTexture(0, d_tex, d_sub_g, sub_g_size);
    //texture <cosets, 1, hipReadModeElementType> d_tex;
    unsigned char *data_cu, *secret_cu;
    int tile_width = 224;
    int remain = secret_size % 7;
    int b_remain = (secret_size-remain)%tile_width;
    dim3 dimBlock(tile_width);
    dim3 dimGrid(secret_size/tile_width);
    printf("secret size + %d\n",secret_size);
    hipMalloc((void**)&data_cu, secret_size+1);
    hipMalloc((void**)&secret_cu, secret_size);
    printf("size: %d\n",encoder.width*encoder.height);
    hipMemcpy(data_cu, encoder.data, encoder.width*encoder.height, hipMemcpyHostToDevice);
    hipMemcpy(secret_cu, bits, secret_size, hipMemcpyHostToDevice);
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    for(int n=0; n<10; n++)
	printf("%d ",encoder.data[n]);
    printf("\n");
    embed<<<dimGrid, dimBlock,0,stream1>>> (data_cu, encoder.data_size, secret_cu, secret_size-remain-b_remain, d_sub_g, 0);
    embed<<<1, b_remain,0,stream2>>> (data_cu, encoder.data_size, secret_cu, b_remain, d_sub_g, secret_size - remain - b_remain);
    //remain_embed(encoder.data, encoder.data_size, bits, secret_size, host_sub_g);
    hipMemcpy(encoder.data, data_cu, encoder.height*encoder.width, hipMemcpyDeviceToHost);
    //unsigned char* test = (unsigned char*)malloc(secret_size*sizeof(unsigned char));
    //hipMemcpy(test, secret_cu, secret_size, hipMemcpyDeviceToHost);
    //char* test_message = (char*)malloc((secret_size/8)*sizeof(char)+1);
    //BitsToString(test,secret_size,test_message);
    //printf("test\n%s\n",test_message);
    for(int n=0; n<10; n++)
	printf("%d ",encoder.data[n]);
    printf("\n");
    printf("\nAfter encode\n");
    OutputFile("photo/encode.bmp", &encoder);
    free(bits);
    free(string);
    hipFree(data_cu);
    hipFree(secret_cu);
    // stego decoder
    struct BmpParser decoder;
    ReadFile("photo/encode.bmp", &decoder);
    char* message = (char*)malloc((secret_size/8)*sizeof(char)+1);
    decode(decoder.data, secret_size, message);
    printf("end\n");
    printf("message = %s\n", message);

    // output txt file
    OutputTxt("message.txt", message);
    free(message);
    return 0;
}
