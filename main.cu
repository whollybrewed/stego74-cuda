#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include "bmp_parser.h"
#include "embed.h"
#include "decoder.h"
int main(int argc, char* argv[])
{
    // bmp reader
    struct BmpParser encoder;
    ReadFile(argv[2], &encoder);
    
    // read message from txt file
    int secret_size = (encoder.width*encoder.height-1);
    int string_size = secret_size/8;
    char* string = (char*)malloc(string_size*sizeof(unsigned char)+1);
    ReadTxt(argv[1], string, string_size);

    // stego grouping
    unsigned char* bits = (unsigned char*)malloc(secret_size*sizeof(unsigned char));
    StringToBits(string, bits);
    cosets *d_sub_g;
    dim3 BlockSize(128);
    dim3 GridSize(1);
    int sub_g_size = 128 * sizeof(cosets);
    hipMalloc((void**)&d_sub_g, sub_g_size);
    grouping<<<GridSize, BlockSize>>>(d_sub_g);

    //stego embed
    unsigned char *data_cu, *secret_cu;
    int tile_width = 224;
    int remain = secret_size % 7;
    int b_remain = (secret_size-remain)%tile_width;
    dim3 dimBlock(tile_width);
    dim3 dimGrid(secret_size/tile_width);
    hipMalloc((void**)&data_cu, secret_size + 1);
    hipMalloc((void**)&secret_cu, secret_size);
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    printf("Embedding...");
    hipMemcpyAsync(data_cu, encoder.data, encoder.width*encoder.height, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(secret_cu, bits, secret_size, hipMemcpyHostToDevice, stream1);
    embed<<<dimGrid, dimBlock,0,stream2>>>
        (data_cu, encoder.data_size, secret_cu, secret_size - remain - b_remain, d_sub_g, 0);
    embed<<<1, b_remain,0,stream1>>>
        (data_cu, encoder.data_size, secret_cu, b_remain, d_sub_g, secret_size - remain - b_remain);
    hipMemcpyAsync(encoder.data, data_cu, encoder.height * encoder.width, hipMemcpyDeviceToHost, stream2);
    OutputFile("photo/encode.bmp", &encoder);
    printf("Output embedded image\n");
    free(bits);
    free(string);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
    hipFree(data_cu);
    hipFree(secret_cu);

    // stego decoder
    struct BmpParser decoder;
    ReadFile("photo/encode.bmp", &decoder);
    char* message = (char*)malloc((secret_size/8)*sizeof(char)+1);
    printf("Decoding...\n");
    decode(decoder.data, secret_size, message);

    // output txt file
    OutputTxt("message.txt", message);
    printf("Output decoded message\n");
    printf("secrets character count: %d\n", strlen(message));
	printf("==========================================================\n\n");
    free(message);
	hipProfilerStop();
    return 0;
}
