#include "hip/hip_runtime.h"
#include "decoder.h"
#include "convert.h"
#include "grouping.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void decodekernel(unsigned char *dp, unsigned char *dl, unsigned char *ds)
{
   	unsigned char matrix_h[3][7] =
 	{
          {1, 1, 0, 1, 1, 0, 0},
          {1, 0, 1, 1, 0, 1, 0},
          {0, 1, 1, 1, 0, 0, 1}
 	};
	int i;
   	int tx = blockIdx.x*blockDim.x+threadIdx.x;
        ds[tx*7] = 0;
        ds[tx*7+1] = 0;
        ds[tx*7+3] = 0;
        for(i=0; i<7; i++){
                dl[tx*7+i] = dp[tx*7+i]&1;
                ds[tx*7] = ds[tx*7]^(matrix_h[0][i]*dl[tx*7+i]);
                ds[tx*7+1] = ds[tx*7+1]^(matrix_h[1][i]*dl[tx*7+i]);
                ds[tx*7+3] = ds[tx*7+3]^(matrix_h[2][i]*dl[tx*7+i]);
        }
        ds[tx*7+2] = dl[tx*7+2];
        ds[tx*7+4] = dl[tx*7+4];
        ds[tx*7+5] = dl[tx*7+5];
        ds[tx*7+6] = dl[tx*7+6];
}

void decode(unsigned char *p, const int secret_size, char* message)
{
	unsigned char matrix_h[3][7] =
	{
	  {1, 1, 0, 1, 1, 0, 0},
	  {1, 0, 1, 1, 0, 1, 0},
	  {0, 1, 1, 1, 0, 0, 1}
	};
	int i, j;
        const int num_secret = secret_size;
        const int num_group = (secret_size)/7;
        const int remain = (secret_size) % 7;
        unsigned char l[num_secret-remain+7];  //LSB
        unsigned char s[num_secret-remain+7];  //secret bits
	int size = num_group*7*sizeof(unsigned char);
	int tilewid = 7;
	unsigned char *dp, *dl, *ds;
        hipMalloc(&dp, size);
        hipMemcpy(dp, p, size, hipMemcpyHostToDevice);
	hipMalloc(&dl, size);
        hipMemcpy(dl, l, size, hipMemcpyHostToDevice);
        hipMalloc(&ds, size);
	dim3 dimBlock(tilewid);
        dim3 dimGrid(num_group/tilewid);
        decodekernel<<<dimGrid, dimBlock>>>(dp, dl, ds);
        printf("Printing final results...\n");
        hipMemcpy(s, ds, size, hipMemcpyDeviceToHost);
	hipFree(dp);
	hipFree(dl);
        hipFree(ds);
        //deal with the remainder
        if(remain>0){
		for(j=num_group*7; j<num_secret; j++)
			l[j] = p[j]&1;
                for(j=num_secret-remain+6; j>=num_secret; j--){
                        l[j] = p[num_secret]&1;
                        p[num_secret] = p[num_secret]>>1;
                }
		s[num_group*7] = 0;
		s[num_group*7+1] = 0;
		s[num_group*7+3] = 0;
                for(i=0; i<7; i++){
                        s[num_group*7] = s[num_group*7]^(matrix_h[0][i]*l[num_secret-remain+i]);
                        s[num_group*7+1] = s[num_group*7+1]^(matrix_h[1][i]*l[num_secret-remain+i]);
                        s[num_group*7+3] = s[num_group*7+3]^(matrix_h[2][i]*l[num_secret-remain+i]);
                }
                s[num_group*7+2] = l[num_group*7+2];
                s[num_group*7+4] = l[num_group*7+4];
                s[num_group*7+5] = l[num_group*7+5];
                s[num_group*7+6] = l[num_group*7+6];
        }
    BitsToString(s, secret_size/8+1, message);
    message[secret_size/8] = '\0';
}
